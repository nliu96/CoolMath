#pragma once
#include <hip/hip_runtime.h>


#include <iostream>

template <typename scalar>
__global__ void vector_add_kernel(const scalar *a, const scalar *b, scalar *c, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        c[i] = a[i] + b[i];
    }
}

template <typename scalar>
inline void vector_add(const scalar *a, const scalar *b, scalar *c, int numElements) {
  int threadsPerBlock = 256;
  int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  vector_add_kernel<<<blocksPerGrid, threadsPerBlock>>>(a, b, c, numElements);
}